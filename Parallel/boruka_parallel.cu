#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string>
#include <iostream>
//#include <string.h>
//#include <assert.h>
//#include <stdlib.h>
#include <hip/hip_runtime.h>
//#include <time.h>
#include <stdlib.h>



#include <fstream>
#include <fstream>
#include <iomanip>
// includes, project
////////////////////////////////////////////////////////////////////////////////
// declarations, forward

#define WIDTH 32 
int total_weight=0;
//extern "C"
// a structure to represent a weighted edge in graph
struct Edge
{
    int src, dest, weight;
};
 
// a structure to represent a connected, undirected
// and weighted graph as a collection of edges.
struct Graph
{
    // V-> Number of vertices, E-> Number of edges
    int V, E;
 
    // graph is represented as an array of edges.
    // Since the graph is undirected, the edge
    // from src to dest is also edge from dest
    // to src. Both are counted as 1 edge here.
    Edge* edge;
};
 
// A structure to represent a subset for union-find
struct subset
{
    int parent;
    int rank;
};
struct Graph* createGraph(int V, int E)
{
    Graph* graph = (Graph*)malloc(sizeof(Graph));
    graph->V = V;
    graph->E = E;
    graph->edge = (Edge *)malloc(E*sizeof(Edge));
    return graph;
}
struct Graph* create_graph(char *filename);
__device__ int find(struct subset subsets[], int i);
__device__ void Union(struct subset subsets[], int x, int y);


__device__ int find(struct subset subsets[], int i)
{
	

    // find root and make root as parent of i
    // (path compression)
    //printf("subsets[i].parent is = %d,i = %d",subsets[i].parent,i);

	for (int k=i;k<100;k++)
	{
	 if (subsets[i].parent == i)
	    return subsets[i].parent;
	else
		continue;
	}
 		//printf("inside kernel\n");
		    return subsets[i].parent;


}
 
// A function that does union of two sets of x and y
// (uses union by rank)
__device__ void Union(struct subset subsets[], int x, int y)
{
    int xroot = find(subsets, x);
    int yroot = find(subsets, y);
 
    // Attach smaller rank tree under root of high
    // rank tree (Union by Rank)
    if (subsets[xroot].rank < subsets[yroot].rank)
        subsets[xroot].parent = yroot;
    else if (subsets[xroot].rank > subsets[yroot].rank)
        subsets[yroot].parent = xroot;
 
    // If ranks are same, then make one as root and
    // increment its rank by one
    else
    {
        subsets[yroot].parent = xroot;
        subsets[xroot].rank++;
    }
}
 
__global__ void find_subsets(struct subset* subsets, Edge* edge, int* cheapest_in, int* cheapest_out, unsigned long long* runtime)
{
	    int tid=threadIdx.x;
	unsigned long long start_time=clock64();
            // Find components (or sets) of two corners
            // of current edge
	   // printf("Kernel Address %u  and tid:%d\n",edge,tid);
		//printf ("edgesrc %d\n", edge[tid].src);
            int set1 = find(subsets, edge[tid].src);
 		//printf("inside kernel\n");
            int set2 = find(subsets, edge[tid].dest);

            // If two corners of current edge belong to
            // same set, ignore current edge
            if (set1 == set2)
               return;

            // Else check if current edge is closer to previous
            // cheapest edges of set1 and set2
           
               if (cheapest_in[set1] == -1 ||
                   edge[cheapest_in[set1]].weight > edge[tid].weight)
                 cheapest_out[set1] = tid;
 
               if (cheapest_in[set1] == -1 ||
                   edge[cheapest_in[set2]].weight > edge[tid].weight)
                 cheapest_out[set2] = tid;

		//printf ("thread id %d\n",tid);

  	unsigned long long stop_time=clock64();
	runtime[tid]=(unsigned long long)(stop_time-start_time);//runtime for each thread

}

 
__global__ void subsets_weight(struct subset* subsets, Edge* edge, int* cheapest_in, int* MSTweight, int* numTrees, unsigned long long* runtime)
{
	    int tid=threadIdx.x;

	unsigned long long start_time=clock64();
            // Check if cheapest for current set exists
            if (cheapest_in[tid] != -1)
            {

                int set1 = find(subsets, edge[cheapest_in[tid]].src);

                int set2 = find(subsets, edge[cheapest_in[tid]].dest);

               // if (set1 != set2)
                 //   continue;

                MSTweight[tid] = edge[cheapest_in[tid]].weight;
	//	printf ("mst in kernel %d\n", MSTweight[tid]);
              //  printf("Edge %d-%d-%d included in MST\n",
                //       edge[cheapest[i]].src, edge[cheapest[i]].dest,
                  //     edge[cheapest[i]].weight);
 
                // Do a union of set1 and set2 and decrease number
                // of trees

                Union(subsets, set1, set2);//--
                numTrees[tid]=0;
		//printf ("inside ker\n");
            }
	else
	{
		MSTweight[tid]=0;
		numTrees[tid]=1;
	}
  
	unsigned long long stop_time=clock64();
	runtime[tid]=(unsigned long long)(stop_time-start_time);//runtime for each thread

}

/*

void boruvkaMST(struct Graph* graph)
{


}	


*/
/**
 * Host main routine
 */
//#define V 100
//#define E 100
   #define EDGE E
   #define VERTEX V
int main(int argc, char *argv[]) 
{

   if (argc < 2) {
    printf("Error: usage: %s <program_file_1> <program_file_2> ...\n",
           argv[0]);
    exit(1);
   }

    printf("boruvka Simulator\n\n");

  //  struct Graph* graph = create_graph(argv[1]);
	

    char *file = argv[1];
    char *line_arr = (char *)malloc(100*sizeof(char));
    std::string line;
    std::ifstream myfile (file);
   int V, E;
    getline(myfile, line);
    strcpy(line_arr, line.c_str());
   // printf("string is %s\n",line_arr);
    char *graph_dims = strtok(line_arr, " ");
    V = atoi(graph_dims);
    //printf("Vertices %d\n",V);
    graph_dims = strtok(NULL, " ");
    E = atoi(graph_dims);
  //  printf("Edges %d\n",E);
    Edge *edge = (Edge *)malloc(2*E*sizeof(Edge));

    struct Graph* g = createGraph(V, 2*E);

 //   Graph g = instantiate_graph(V, 2 * E);

    int offset_count = 0;
    int temp;
    
    while(getline (myfile, line))
    {
		int src=0, dest=0;

        strcpy(line_arr, line.c_str());
      //  printf("string is %s\n",line_arr);
        char* s = strtok(line_arr, " ");
        src = atoi(s);
        s = strtok(NULL, " ");
        dest = atoi(s);
        //printf("dest is %d\n",dest);
        s = strtok(NULL, " ");
        int weight = atoi(s);
        total_weight+=weight;
        //printf("weight is %d\n",weight);
        g->edge[offset_count].src = src;
        g->edge[offset_count].dest = dest;
        g->edge[offset_count].weight = weight;
		
        offset_count++;
       // printf ("after ifstream %d\n", offset_count);

    }
    
   // printf ("after graph generation\n");
        printf ("total weight %d\n", total_weight);

    myfile.close();
    free(line_arr);
    	//printf ("eof\n");



	hipError_t err = hipSuccess;
    // Get data of given graph
   // int V = graph->V, E = graph->E;
    edge = g->edge;	
 
	//printf ("Edge details %d\n", edge[5].weight);
    // Allocate memory for creating V subsets.
  //  struct subset *subsets = new subset[V];
    
    struct subset *subsets = (subset *)malloc(V*sizeof(subset));
 
    // An array to store index of the cheapest edge of
    // subset.  The stored index for indexing array 'edge[]'
  //  int *cheapest = new int[V];
    int *cheapest = (int *)malloc(V*sizeof(int));
 

    // Create V subsets with single elements
    for (int v = 0; v < V; ++v)//--
    {
        subsets[v].parent = v;
        subsets[v].rank = 0;
        cheapest[v] = -1;
    }
    
//printf ("inside\n");
 
    // Initially there are V different trees.
    // Finally there will be one tree that will be MST
    int numTrees = V;
    int MSTweight = 0;
    unsigned long long net_runtime=0;//stores the total execution time
   subset *d_subsets;//=(subset *)malloc(V*sizeof(subset));
	//printf ("inside1\n");	
   err = hipMalloc((void**)&d_subsets, V*sizeof(subset)); // TODO: Allocate context in GPU device memory
//printf ("inside2\n"); 
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device subset data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


//printf ("inside1\n");	
    Edge *d_edge;//=(Edge *)malloc(sizeof(Edge));

//printf ("inside2\n"); 
    err = hipMalloc((void**)&d_edge, 2*E*sizeof(Edge)); // TODO: Allocate context in GPU device memory 
//printf ("inside2\n"); 
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device edge data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    Edge *d_graph;//=(Edge *)malloc(sizeof(Edge));

//printf ("inside2\n"); 
/*
    err = hipMalloc((void**)&d_graph, E*sizeof(Graph)); // TODO: Allocate context in GPU device memory 
//printf ("inside2\n"); 
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device graph data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
*/


    int *d_cheapest;// = (int *)malloc(100*sizeof(int));

    err = hipMalloc((void**)&d_cheapest, V*sizeof(int)); // TODO: Allocate context in GPU device memory 
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device cheapest data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
// printf ("inside2\n");
   int* d_cheapest_out;// = (int*)malloc(100*sizeof(int));

    err = hipMalloc((void**)&d_cheapest_out, V*sizeof(int)); // TODO: Allocate context in GPU device memory 
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device cheapest out data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


//printf("Here\n");
/*-------------runtime related memory allocation----------*/
    unsigned long long* d_runtime;
    int r_size = EDGE*VERTEX*sizeof(unsigned long long);
    unsigned long long* runtime = (unsigned long long*)malloc(r_size);
    memset(runtime, 0, r_size);
    hipMalloc((void**)&d_runtime, r_size);
/*-------------------------xxxxxxxxx-----------------------*/


    int* d_MSTweight;
    int mst_size = VERTEX*sizeof(int);
    int* mst_weight = (int*)malloc(mst_size);
    memset(mst_weight, 0, mst_size);
    hipMalloc((void**)&d_MSTweight, mst_size);
/*---------------------------------xxxxxxxxxx----------------*/

    int* d_numTrees;
    int trees_size = VERTEX*sizeof(int);
    int* num_of_trees = (int*)malloc(trees_size);
    memset(num_of_trees, 0, trees_size);
    hipMalloc((void**)&d_numTrees, trees_size);

/*------------------xxxxxxxxxxxxxx--------------------*/

// int EDGE = E;
// int VERTEX = V;

    // Keep combining components (or sets) until all
    // compnentes are not combined into single MST.
 //  printf("Number of trees %d\n",numTrees);
    while (numTrees > 1)//--
    {

        // Traverse through all edges and update
        // cheapest of every component
    //    printf("First for loop1");

    err = hipMemcpy(d_subsets, subsets, V*sizeof(subset), hipMemcpyHostToDevice);// TODO: Copy the input/updated context to GPU
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy subset data from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_edge, edge, 2*E*sizeof(Edge), hipMemcpyHostToDevice);// TODO: Copy the input/updated context to GPU
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy edge data from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }



    err = hipMemcpy(d_cheapest, cheapest, V*sizeof(int), hipMemcpyHostToDevice);// TODO: Copy the input/updated context to GPU
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy cheapest data from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
/*
    err = hipMemcpy(d_numTrees, numTrees, sizeof(int), hipMemcpyHostToDevice);// TODO: Copy the input/updated context to GPU
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy numTrees data from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
*/



    dim3 dimGrid1(EDGE,1, 1);
    dim3 dimBlock1(1, 1, 1);
	//printf("Address %u\n",d_edge);

// Call the kernel function
    find_subsets<<<dimBlock1,dimGrid1>>>(d_subsets, d_edge, d_cheapest, d_cheapest_out, d_runtime);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Kernel-1 execution failed (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();

    //printf("Copy between kernel data from the CUDA device to the host memory\n");//copying the updated context from GPU to CPU
    err = hipMemcpy(cheapest,d_cheapest_out, V*sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy between kernel data from device to host (error code %s)!\n", hipGetErrorString(err));
     }

    err = hipMemcpy(d_cheapest, cheapest, V*sizeof(int), hipMemcpyHostToDevice);// TODO: Copy the input/updated context to GPU
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy cheapest data second time from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipMemcpy(runtime, d_runtime, r_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    unsigned long long elapsed_time_EDGE = 0;
    for(int i = 0; i < EDGE; i++)
        if(elapsed_time_EDGE < runtime[i])
            elapsed_time_EDGE = runtime[i];//highest execution time among all the simultaneously running threads
    net_runtime += elapsed_time_EDGE;// calculates the total execution time, each time when the kernel is executed


    dim3 dimGrid2(VERTEX,1, 1);
    dim3 dimBlock2(1, 1, 1);
    // Call the kernel function
    subsets_weight<<<dimBlock2,dimGrid2>>>(d_subsets, d_edge, d_cheapest, d_MSTweight, d_numTrees, d_runtime);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Kernel-2 execution failed (error code %s)\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();

    err = hipMemcpy(subsets, d_subsets, V*sizeof(subset), hipMemcpyDeviceToHost);// TODO: Copy the input/updated context to GPU
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy subset data from device to host device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
   hipMemcpy(runtime, d_runtime, r_size, hipMemcpyDeviceToHost);//--
    hipDeviceSynchronize();
    
    unsigned long long elapsed_time_VER = 0;
    for(int j = 0; j < VERTEX; j++)
        if(elapsed_time_VER < runtime[j])
            elapsed_time_VER = runtime[j];//highest execution time among all the simultaneously running threads
    net_runtime += elapsed_time_VER;// calculates the total execution time, each time when the kernel is executed

    hipMemcpy(num_of_trees, d_numTrees, trees_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    

    for(int k = 0; k < VERTEX; k++)
	numTrees += num_of_trees[k];
    numTrees = numTrees/VERTEX;
    //printf("num trees %d\n", numTrees);
//numTrees--;
   //printf ("num of trees %d\n", numTrees);
  // printf ("mst weight %d\n", d_MSTweight);
    hipMemcpy(mst_weight, d_MSTweight, mst_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
   int temp_weight=0;
    for(int n = 0; n < VERTEX; n++)
	//if (temp_weight<mst_weight[n])
	//	temp_weight=mst_weight[n];
	MSTweight += mst_weight[n];
    }
 hipFree(d_subsets);
 hipFree(d_edge);
 hipFree(d_cheapest);
 hipFree(d_cheapest_out);
   // printf("Weight of MST is %d\n", MSTweight);
    printf("total run time in nanoseconds on GPU %d\n",int(net_runtime*8.17)/64);
    return;

	return 0;
}







///--------working function--------///

/*
struct Graph* create_graph(char *filename)
{
    char *file = filename;
//(Edge *)malloc(sizeof(Edge))
    char *line_arr = (char *)malloc(100*sizeof(char));
    std::string line;
    std::ifstream myfile (file);
   // int V, E;
    getline(myfile, line);
    strcpy(line_arr, line.c_str());
    printf("string is %s\n",line_arr);
    char *graph_dims = strtok(line_arr, " ");
    //V = atoi(graph_dims);
    printf("Vertices %d\n",V);
    graph_dims = strtok(NULL, " ");
    //E = atoi(graph_dims);
    printf("Edges %d\n",E);

    struct Graph* g = createGraph(V, 2*E);

 //   Graph g = instantiate_graph(V, 2 * E);

    int offset_count = 0;
    int temp;
    
    while(getline (myfile, line))
    {
		int src=0, dest=0;

        strcpy(line_arr, line.c_str());
        printf("string is %s\n",line_arr);
        char* s = strtok(line_arr, " ");
        src = atoi(s);
        s = strtok(NULL, " ");
        dest = atoi(s);
        printf("dest is %d\n",dest);
        s = strtok(NULL, " ");
        int weight = atoi(s);
        total_weight+=weight;
        printf("weight is %d\n",weight);
        g->edge[offset_count].src = src;
        g->edge[offset_count].dest = dest;
        g->edge[offset_count].weight = weight;
		
        offset_count++;
        printf ("after ifstream %d\n", offset_count);

    }
    
    printf ("after graph generation\n");
        printf ("total weight %d\n", total_weight);

    myfile.close();
    free(line_arr);
    	printf ("eof\n");

    return g;
}*/
